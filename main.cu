
#include <hip/hip_runtime.h>
#include <cstdio>


int getThreadNum()
{
	hipDeviceProp_t prop;
	int count;
	hipGetDeviceCount(&count);
	printf("gpu num %d\n", count);

	hipGetDeviceProperties(&prop, 0);
	printf("max thread num : %d\n", prop.maxThreadsPerBlock);
	printf("grid dimensions : %d %d %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	return prop.maxThreadsPerBlock;
}

__global__ void conv (float* img, float* kernel, float* result, int width, int
		height, int channel, int kernelSize)
	
{
	int ti = threadIdx.x;
	int bi = blockIdx.x;
	// int id = (bi * threadNum + ti);
	int id = (bi * blockDim.x + ti);
	if (id >= width * height * channel) return;
	
	int row = id / width;
	int col = id % width;

	for ( int k = 0;k < channel;++k ) {
		for ( int i = 0;i < kernelSize;++i ) {
			for ( int j = 0;j < kernelSize;++j ) {
				float imgValue = 0.0;
				int curRow = row - kernelSize >> 1 + i;
				int curCol = col - kernelSize >> 1 + j;
				if ( curRow < 0 || curCol < 0 || curRow >= height || curCol >= width) {
						continue;
				} else {
					imgValue = img[curRow * width + curCol];
				}
				result[id] += kernel[k * kernelSize * kernelSize + i * kernelSize + j] * imgValue;
			}
		}
	}
//	for ( int i = 0;i < kernelSize; ++i ) {
//		for ( int j = 0;j < kernelSize; ++j ) {
//			float imgValue = 0.0;
//			int curRow = row - kernelSize / 2 + i;
//			int curCol = col - kernelSize / 2 + j;
//			if (curRow < 0 || curCol < 0 || curRow >= height || curCol >= width) {
//				continue;	
//			} else {
//				imgValue = img[curRow * width + curCol];
//			}
//			result[id] += kernel[i * kernelSize + j] * imgValue;
//		}
//	}  
}


int main(void)
{
	int width = 1920;
	int height = 1080;
	int inChannel = 1;
	int outChannel = 8;

	float* img = new float[width * height];

	for ( int row = 0;row < height;++row ) {
		for (int col = 0;col < width;++col ) {
			img[col + row * width] = (col + row) % 256;
		}

	}		
	int kernelSize = 5;
	float* kernel = new float[outChannel * kernelSize * kernelSize];

	for ( int i = 0;i < outChannel * kernelSize * kernelSize; ++i) {
		kernel[i] = i % kernelSize - 1;
	}

	float* imgGpu;
	float* kernelGpu;
	float* resultGpu;

	hipMalloc((void**)&imgGpu, inChannel * width * height * sizeof(float));
	hipMalloc((void**)&kernelGpu, outChannel * kernelSize * kernelSize * sizeof(float));
	hipMalloc((void**)&resultGpu, outChannel * width * height * sizeof(float));


	hipMemcpy(imgGpu, img, inChannel * width * height * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(kernelGpu, kernel, outChannel * kernelSize * kernelSize * sizeof(float), hipMemcpyHostToDevice);


	int threadNum = getThreadNum();
	int blockNum = (width * height - 0.5) / threadNum + 1;
	conv<<<blockNum, threadNum>>>(imgGpu, kernelGpu, resultGpu, width, height,
			outChannel, kernelSize);
	

	float* result = new float[outChannel * width * height];
	hipMemcpy(result, resultGpu, outChannel * width * height * sizeof(float), hipMemcpyDeviceToHost);
	
	

	for ( int i = 0;i < 10;++i ) {
		for ( int j = 0;j < 10;++j ) {
			printf("%2.0f ", img[i * width + j]);
		}
		printf("\n");
	} 
	printf("\n");

	for ( int k = 0;k < 2;++k ) {
	for ( int i = 0;i < kernelSize;++i ) {
		for ( int j = 0;j < kernelSize;++j ) {
			printf("%2.0f ", kernel[k * kernelSize * kernelSize + i * kernelSize + j]);
		}
		printf("\n");
	}
	printf("\n\n");
	}

	for ( int k = 0;k < 2;++k ){ 
	for ( int i = 0;i < 10;++i ) {
		for ( int j = 0;j < 10;++j ) {
			printf("%2.0f ", result[k * height * width + i * width + j]);
		}
		printf("\n");
	}	
	printf("\n\n");
	}

	return 0;
}
